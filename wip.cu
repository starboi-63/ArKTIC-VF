#include "hip/hip_runtime.h"
extern "C" __global__ void kernel_Syn_updateOutput(
        const int n,
        const float* input,
        const float* weight, 
        const float* offset_x,
        const float* offset_y,
        float* output
) 
{ 
    for (int intIndex = (blockIdx.x * blockDim.x) + threadIdx.x; intIndex < n; intIndex += blockDim.x * gridDim.x) {
        float dblOutput = 0.0;

        const int intSample = ( intIndex / SIZE_3(output) / SIZE_2(output) / SIZE_1(output) ) % SIZE_0(output);
        const int intDepth  = ( intIndex / SIZE_3(output) / SIZE_2(output)                  ) % SIZE_1(output);
        const int y         = ( intIndex / SIZE_3(output)                                   ) % SIZE_2(output);
        const int x         = ( intIndex                                                    ) % SIZE_3(output);
    
        for (int row = 0; row < F_SIZE; row += 1) {
            for (int col = 0; col < F_SIZE; col += 1) {
                float w         = VALUE_4(weight, intSample, row*F_SIZE+col, y, x);
                float alpha     = VALUE_4(offset_x, intSample, row*F_SIZE+col, y, x);
                float beta      = VALUE_4(offset_y, intSample, row*F_SIZE+col, y, x);
                int intAlpha    = (int)alpha;
                int intBeta     = (int)beta;

                int bottom = CLAMP(y + row*DILATION + intAlpha, SIZE_2(input) - 1);
                int left = CLAMP(x + col*DILATION + intBeta, SIZE_3(input) - 1);
                int top = CLAMP(y + row*DILATION + intAlpha + 1, SIZE_2(input) - 1);
                int right = CLAMP(x + col*DILATION + intBeta + 1, SIZE_3(input) - 1);

                float alphaTrunc = alpha - (float)intAlpha;
                float betaTrunc = beta - (float)intBeta;

                dblOutput += w * (
                    VALUE_4(input, intSample, intDepth, bottom, left)*(1 - alphaTrunc)*(1 - betaTrunc) + 
                    VALUE_4(input, intSample, intDepth, top, left)*alphaTrunc*(1 - betaTrunc) + 
                    VALUE_4(input, intSample, intDepth, bottom, right)*(1 - alphaTrunc)*betaTrunc + 
                    VALUE_4(input, intSample, intDepth, top, right)*alphaTrunc*betaTrunc
                );
            }
        }

        output[intIndex] = dblOutput;
    } 
}


extern "C" __global__ void kernel_AdaCoF_updateGradWeight(
    const int n,
    const float* gradLoss,
    const float* input,
    const float* offset_y,
    const float* offset_x,
    float* gradWeight
) 
{ 
    for (int intIndex = (blockIdx.x * blockDim.x) + threadIdx.x; intIndex < n; intIndex += blockDim.x * gridDim.x) {
        float floatOutput = 0.0;

        const int intSample  = ( intIndex / SIZE_3(gradWeight) / SIZE_2(gradWeight) / SIZE_1(gradWeight) ) % SIZE_0(gradWeight);
        const int intDepth   = ( intIndex / SIZE_3(gradWeight) / SIZE_2(gradWeight)                      ) % SIZE_1(gradWeight);
        const int y          = ( intIndex / SIZE_3(gradWeight)                                           ) % SIZE_2(gradWeight);
        const int x          = ( intIndex                                                                ) % SIZE_3(gradWeight);

        int row = intDepth / F_SIZE;
        int col = intDepth % F_SIZE;

        for (int depth = 0; depth < 3; depth++) {
            float delta     = VALUE_4(gradLoss, intSample, depth, y, x);
            float alpha     = VALUE_4(offset_y, intSample, row*F_SIZE+col, y, x);
            float beta      = VALUE_4(offset_x, intSample, row*F_SIZE+col, y, x);
            int intAlpha    = (int)alpha;
            int intBeta     = (int)beta;

            int bottom = CLAMP(y + row*DILATION + intAlpha, SIZE_2(input) - 1);
            int left = CLAMP(x + col*DILATION + intBeta, SIZE_3(input) - 1);
            int top = CLAMP(y + row*DILATION + intAlpha + 1, SIZE_2(input) - 1);
            int right = CLAMP(x + col*DILATION + intBeta + 1, SIZE_3(input) - 1);

            float alphaTrunc = alpha - (float)intAlpha;
            float betaTrunc = beta - (float)intBeta;
            
            floatOutput += delta * (
                VALUE_4(input, intSample, depth, bottom, left)*(1 - alphaTrunc)*(1 - betaTrunc) + 
                VALUE_4(input, intSample, depth, top, left)*alphaTrunc*(1 - betaTrunc) + 
                VALUE_4(input, intSample, depth, bottom, right)*(1 - alphaTrunc)*betaTrunc + 
                VALUE_4(input, intSample, depth, top, right)*alphaTrunc*betaTrunc
            );
        }

        gradWeight[intIndex] = floatOutput;
    } 
}

